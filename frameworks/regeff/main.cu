#include "hip/hip_runtime.h"
#include <iostream>

#include "Instance.cuh"
#include "UtilityFunctions.cuh"

template <typename MemoryManager>
__global__ void d_testFunctions(MemoryManager memory_manager)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid > 0)
		return;

	int* test_array = reinterpret_cast<int*>(memory_manager.malloc(sizeof(int) * 16));

	for(int i = 0; i < 16; ++i)
	{
		test_array[i] = i;
	}

	memory_manager.free(test_array);

	printf("It worked!\n");

	return;
}

int main(int argc, char* argv[])
{
	std::cout << "Simple RegEff Testcase\n";
	
	{
		MemoryManagerRegEff<RegEffVariants::CudaMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase CudaMalloc done!\n");
		printf("###############################################\n");
	}

	{
		MemoryManagerRegEff<RegEffVariants::AtomicMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase AtomicMalloc done!\n");
		printf("###############################################\n");
	}

	{
		MemoryManagerRegEff<RegEffVariants::AWMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase AWMalloc done!\n");
		printf("###############################################\n");
	}

	{
		MemoryManagerRegEff<RegEffVariants::CMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase CMalloc done!\n");
		printf("###############################################\n");
	}

	{
		MemoryManagerRegEff<RegEffVariants::CFMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase CFMalloc done!\n");
		printf("###############################################\n");
	}

	{
		MemoryManagerRegEff<RegEffVariants::CMMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase CMMalloc done!\n");
		printf("###############################################\n");
	}

	{
		MemoryManagerRegEff<RegEffVariants::CFMMalloc> memory_manager;

		d_testFunctions <<<1,1>>>(memory_manager);

		CHECK_ERROR(hipDeviceSynchronize());

		printf("Testcase CFMMalloc done!\n");
		printf("###############################################\n");
	}


	return 0;
}

