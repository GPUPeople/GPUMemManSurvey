#include "hip/hip_runtime.h"
#include <iostream>

#include "Instance.cuh"

template <typename MemoryManager>
__global__ void d_testFunctions(MemoryManager memory_manager)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid > 0)
		return;

	int* test_array = reinterpret_cast<int*>(memory_manager.malloc(sizeof(int) * 16));

	for(int i = 0; i < 16; ++i)
	{
		test_array[i] = i;
	}

	memory_manager.free(test_array);

	printf("It worked!\n");

	return;
}

int main(int argc, char* argv[])
{
	std::cout << "Simple ScatterAlloc Testcase\n";

	MemoryManagerScatterAlloc memory_manager;

	d_testFunctions <<<1,1>>>(memory_manager);

	hipDeviceSynchronize();

	printf("Testcase done!\n");

	return 0;
}