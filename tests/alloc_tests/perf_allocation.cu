#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>

#include "UtilityFunctions.cuh"
#include "PerformanceMeasure.cuh"
#include "DevicePerformanceMeasure.cuh"

// ########################
#ifdef TEST_CUDA
#include "cuda/Instance.cuh"
#elif TEST_HALLOC
#include "halloc/Instance.cuh"
#elif TEST_SCATTERALLOC
#include "scatteralloc/Instance.cuh"
#elif TEST_OUROBOROS
#include "ouroboros/Instance.cuh"
#elif TEST_FDG
#include "fdg/Instance.cuh"
#elif TEST_REGEFF
#include "regeff/Instance.cuh"
#endif

template <typename MemoryManagerType, bool warp_based>
__global__ void d_testAllocation(MemoryManagerType mm, int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid{0};
	if(warp_based)
	{
		tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
		if(threadIdx.x % 32 != 0)
			return;
	}
	else
	{
		tid = threadIdx.x + blockIdx.x * blockDim.x;
	}
	if(tid >= num_allocations)
		return;

	verification_ptr[tid] = reinterpret_cast<int*>(mm.malloc(allocation_size));
}

template <typename MemoryManagerType>
__global__ void d_testAllocation(MemoryManagerType mm, int** verification_ptr, int num_allocations, int allocation_size, DevicePerfMeasure::Type* timing)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	DevicePerf perf_measure;
	
	// Start Measure
	perf_measure.startThreadMeasure();

	auto ptr = reinterpret_cast<int*>(mm.malloc(allocation_size));
	
	// Stop Measure
	timing[tid] = perf_measure.stopThreadMeasure();

	verification_ptr[tid] = ptr;
}

template <typename MemoryManagerType, bool warp_based>
__global__ void d_testFree(MemoryManagerType mm, int** verification_ptr, int num_allocations)
{
	int tid{0};
	if(warp_based)
	{
		tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
		if(threadIdx.x % 32 != 0)
			return;
	}
	else
	{
		tid = threadIdx.x + blockIdx.x * blockDim.x;
	}
	if(tid >= num_allocations)
		return;

	mm.free(verification_ptr[tid]);
}

template <typename MemoryManagerType>
__global__ void d_testFree(MemoryManagerType mm, int** verification_ptr, int num_allocations, DevicePerfMeasure::Type* timing)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	DevicePerf perf_measure;

	// Start Measure
	perf_measure.startThreadMeasure();

	mm.free(verification_ptr[tid]);

	// Stop Measure
	timing[tid] = perf_measure.stopThreadMeasure();
}

__global__ void d_testWriteToMemory(int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		ptr[i] = tid;
	}
}

__global__ void d_testReadFromMemory(int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		if(ptr[i] != tid)
		{
			printf("%d | We got a wrong value here! %d vs %d\n", tid, ptr[i], tid);
			__trap();
		}
	}
}

int main(int argc, char* argv[])
{
	// Usage: <num_allocations> <size_of_allocation_in_byte> <num_iterations> <onDeviceMeasure> <warp-based> <generateoutput> <free_memory> <initial_path>
	unsigned int num_allocations{10000};
	unsigned int allocation_size_byte{8192};
	int num_iterations {100};
	bool warp_based{false};
	bool onDeviceMeasure{false};
	bool print_output{true};
	bool generate_output{false};
	bool write_header{false};
	bool free_memory{true};
	std::string initial_path{"../results/tmp/"};
	if(argc >= 2)
	{
		num_allocations = atoi(argv[1]);
		if(argc >= 3)
		{
			allocation_size_byte = atoi(argv[2]);
			if(argc >= 4)
			{
				num_iterations = atoi(argv[3]);
				if(argc >= 5)
				{
					onDeviceMeasure = static_cast<bool>(atoi(argv[4]));
					if(argc >= 6)
					{
						warp_based = static_cast<bool>(atoi(argv[5]));
						if(onDeviceMeasure && warp_based)
						{
							std::cout << "OnDeviceMeasure and warp-based not possible at the same!" << std::endl;
							exit(-1);
						}
						if(argc >= 7)
						{
							generate_output = static_cast<bool>(atoi(argv[6]));
							if(argc >= 8)
							{
								write_header = static_cast<bool>(atoi(argv[7]));
								if(argc >= 9)
								{
									free_memory = static_cast<bool>(atoi(argv[8]));
									if(argc >= 9)
									{
										initial_path = std::string(argv[9]);
									}
								}
							}
						}
					}
				}
			}
		}
	}
	allocation_size_byte = Utils::alignment(allocation_size_byte, sizeof(int));
	if(print_output)
		std::cout << "Number of Allocations: " << num_allocations << " | Allocation Size: " << allocation_size_byte << std::endl;

	int device{0};
	hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);

#ifdef TEST_CUDA
	std::cout << "--- CUDA ---\n";
	MemoryManagerCUDA memory_manager;
	std::string mem_name("CUDA");
#elif TEST_HALLOC
	std::cout << "--- Halloc ---\n";
	MemoryManagerHalloc memory_manager;
	std::string mem_name("Halloc");
#elif TEST_SCATTERALLOC
	std::cout << "--- ScatterAlloc ---\n";
	MemoryManagerScatterAlloc memory_manager;
	std::string mem_name("ScatterAlloc");
#elif TEST_OUROBOROS
	std::cout << "--- Ouroboros ---";
	#ifdef TEST_PAGES
	#ifdef TEST_VIRTUALIZED_ARRAY
	std::cout << " Page --- Virtualized Array ---\n";
	MemoryManagerOuroboros<OuroVAPQ> memory_manager;
	std::string mem_name("Ouroboros-P-VA");
	#elif TEST_VIRTUALIZED_LIST
	std::cout << " Page --- Virtualized List ---\n";
	MemoryManagerOuroboros<OuroVLPQ> memory_manager;
	std::string mem_name("Ouroboros-P-VL");
	#else
	std::cout << " Page --- Standard ---\n";
	MemoryManagerOuroboros<OuroPQ> memory_manager;
	std::string mem_name("Ouroboros-P-S");
	#endif
	#endif
	#ifdef TEST_CHUNKS
	#ifdef TEST_VIRTUALIZED_ARRAY
	std::cout << " Chunk --- Virtualized Array ---\n";
	MemoryManagerOuroboros<OuroVACQ> memory_manager;
	std::string mem_name("Ouroboros-C-VA");
	#elif TEST_VIRTUALIZED_LIST
	std::cout << " Chunk --- Virtualized List ---\n";
	MemoryManagerOuroboros<OuroVLCQ> memory_manager;
	std::string mem_name("Ouroboros-C-VL");
	#else
	std::cout << " Chunk --- Standard ---\n";
	MemoryManagerOuroboros<OuroCQ> memory_manager;
	std::string mem_name("Ouroboros-C-S");
	#endif
	#endif
#elif TEST_FDG
	std::cout << "--- FDGMalloc ---\n";
	MemoryManagerFDG memory_manager;
	std::string mem_name("FDGMalloc");
#elif TEST_REGEFF
	std::cout << "--- RegEff ---";
	#ifdef TEST_ATOMIC
	std::cout << " Atomic\n";
	MemoryManagerRegEff<RegEffVariants::AtomicMalloc> memory_manager;
	std::string mem_name("RegEff-A");
	#elif TEST_ATOMIC_WRAP
	std::cout << " Atomic Wrap\n";
	MemoryManagerRegEff<RegEffVariants::AWMalloc> memory_manager;
	std::string mem_name("RegEff-AW");
	#elif TEST_CIRCULAR
	std::cout << " Circular\n";
	MemoryManagerRegEff<RegEffVariants::CMalloc> memory_manager;
	std::string mem_name("RegEff-C");
	#elif TEST_CIRCULAR_FUSED
	std::cout << " Circular Fused\n";
	MemoryManagerRegEff<RegEffVariants::CFMalloc> memory_manager;
	std::string mem_name("RegEff-CF");
	#elif TEST_CIRCULAR_MULTI
	std::cout << " Circular Multi\n";
	MemoryManagerRegEff<RegEffVariants::CMMalloc> memory_manager;
	std::string mem_name("RegEff-CM");
	#elif TEST_CIRCULAR_FUSED_MULTI
	std::cout << " Circular Fused Multi\n";
	MemoryManagerRegEff<RegEffVariants::CFMMalloc> memory_manager;
	std::string mem_name("RegEff-CFM");
	#endif
#endif

	int** d_memory{nullptr};
	CHECK_ERROR(hipMalloc(&d_memory, sizeof(int*) * num_allocations));

	std::ofstream results_alloc, results_free;
	if(generate_output)
	{
		results_alloc.open((initial_path + std::string("alloc_") + prop.name  + "_" + mem_name + "_" + std::to_string(num_allocations) + ".csv").c_str(), std::ios_base::app);
		results_free.open((initial_path + std::string("free_") + prop.name + "_" + mem_name + "_" + std::to_string(num_allocations) + ".csv").c_str(), std::ios_base::app);
		if(write_header)
		{
			results_alloc << "AllocationSize (in Byte), mean, std-dev, min, max, median";
			results_free << "AllocationSize (in Byte), mean, std-dev, min, max, median";
		}
		results_alloc << "\n" << allocation_size_byte << ",";
		results_free << "\n" << allocation_size_byte << ",";
	}

	int blockSize {256};
	int gridSize {Utils::divup<int>(num_allocations, blockSize)};

	PerfMeasure timing_allocation;
	PerfMeasure timing_free;

	DevicePerfMeasure per_thread_timing_allocation(num_allocations, num_iterations);
	DevicePerfMeasure per_thread_timing_free(num_allocations, num_iterations);

	for(auto i = 0; i < num_iterations; ++i)
	{
		if(onDeviceMeasure)
		{
			d_testAllocation <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations, allocation_size_byte, per_thread_timing_allocation.getDevicePtr());
			CHECK_ERROR(hipDeviceSynchronize());
			per_thread_timing_allocation.acceptResultsFromDevice();
		}
		else
		{
			timing_allocation.startMeasurement();
			if(warp_based)
				d_testAllocation <decltype(memory_manager), true> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations, allocation_size_byte);
			else
				d_testAllocation <decltype(memory_manager), false> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations, allocation_size_byte);
			timing_allocation.stopMeasurement();
			CHECK_ERROR(hipDeviceSynchronize());
		}

		d_testWriteToMemory<<<gridSize, blockSize>>>(d_memory, num_allocations, allocation_size_byte);

		CHECK_ERROR(hipDeviceSynchronize());

		d_testReadFromMemory<<<gridSize, blockSize>>>(d_memory, num_allocations, allocation_size_byte);

		CHECK_ERROR(hipDeviceSynchronize());

		if(free_memory)
		{
			if(onDeviceMeasure)
			{
				d_testFree <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations, per_thread_timing_free.getDevicePtr());
				CHECK_ERROR(hipDeviceSynchronize());
				per_thread_timing_free.acceptResultsFromDevice();
			}
			else
			{
				timing_free.startMeasurement();
				if(warp_based)
					d_testFree <decltype(memory_manager), true> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations);
				else
					d_testFree <decltype(memory_manager), false> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations);
				timing_free.stopMeasurement();
				CHECK_ERROR(hipDeviceSynchronize());
			}
		}
	}

	if(onDeviceMeasure)
	{
		auto alloc_result = per_thread_timing_allocation.generateResult();
		auto free_result = per_thread_timing_free.generateResult();

		if(print_output)
		{
			std::cout << "Timing Allocation: Mean:" << alloc_result.mean_ << "cycles | Median: " << alloc_result.median_ << " ms" << std::endl;
			std::cout << "Timing       Free: Mean:" << free_result.mean_ << "cycles | Median: " << free_result.median_ << " ms" << std::endl;
		}
		if(generate_output)
		{
			results_alloc << alloc_result.mean_ << "," << alloc_result.std_dev_ << "," << alloc_result.min_ << "," << alloc_result.max_ << "," << alloc_result.median_;
			results_free << free_result.mean_ << "," << free_result.std_dev_ << "," << free_result.min_ << "," << free_result.max_ << "," << free_result.median_;
		}
	}
	else
	{
		auto alloc_result = timing_allocation.generateResult();
		auto free_result = timing_free.generateResult();
		if(print_output)
		{
			std::cout << "Timing Allocation: Mean:" << alloc_result.mean_ << "ms" << std::endl;// " | Median: " << alloc_result.median_ << " ms" << std::endl;
			std::cout << "Timing       Free: Mean:" << free_result.mean_ << "ms" << std::endl;// "  | Median: " << free_result.median_ << " ms" << std::endl;
		}
		if(generate_output)
		{
			results_alloc << alloc_result.mean_ << "," << alloc_result.std_dev_ << "," << alloc_result.min_ << "," << alloc_result.max_ << "," << alloc_result.median_;
			results_free << free_result.mean_ << "," << free_result.std_dev_ << "," << alloc_result.min_ << "," << alloc_result.max_ << "," << free_result.median_;
		}
	}
	
	return 0;
}