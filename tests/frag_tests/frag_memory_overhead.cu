#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <utility>
#include <algorithm> 

#include "UtilityFunctions.cuh"

// ########################
#ifdef TEST_CUDA
#include "cuda/Instance.cuh"
#elif TEST_HALLOC
#include "halloc/Instance.cuh"
#elif TEST_XMALLOC
#include "xmalloc/Instance.cuh"
#elif TEST_SCATTERALLOC
#include "scatteralloc/Instance.cuh"
#elif TEST_OUROBOROS
#include "ouroboros/Instance.cuh"
#elif TEST_FDG
#include "fdg/Instance.cuh"
#elif TEST_REGEFF
#include "regeff/Instance.cuh"
#endif

template <typename MemoryManagerType, bool warp_based>
__global__ void d_testAllocation(MemoryManagerType mm, int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid{0};
	if(warp_based)
	{
		tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
		if(threadIdx.x % 32 != 0)
			return;
	}
	else
	{
		tid = threadIdx.x + blockIdx.x * blockDim.x;
	}
	if(tid >= num_allocations)
		return;

	verification_ptr[tid] = reinterpret_cast<int*>(mm.malloc(allocation_size));
}

template <typename MemoryManagerType, bool warp_based>
__global__ void d_testFree(MemoryManagerType mm, int** verification_ptr, int num_allocations)
{
	int tid{0};
	if(warp_based)
	{
		tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
		if(threadIdx.x % 32 != 0)
			return;
	}
	else
	{
		tid = threadIdx.x + blockIdx.x * blockDim.x;
	}
	if(tid >= num_allocations)
		return;

	mm.free(verification_ptr[tid]);
}

int main(int argc, char* argv[])
{
	// Usage: num_allocations size_of_allocation_in_byte print_output
	unsigned int num_allocations{10000};
	unsigned int allocation_size_byte{16};
	int num_iterations {25};
	bool warp_based{false};
	bool print_output{true};
	bool free_memory{true};
	if(argc >= 2)
	{
		num_allocations = atoi(argv[1]);
		if(argc >= 3)
		{
			allocation_size_byte = atoi(argv[2]);
			if(argc >= 4)
			{
				num_iterations = atoi(argv[3]);
				if(argc >= 5)
				{
					warp_based = static_cast<bool>(atoi(argv[4]));
					if(argc >= 6)
					{
						print_output = static_cast<bool>(atoi(argv[5]));
						if(argc >= 7)
							free_memory = static_cast<bool>(atoi(argv[6]));
					}
				}
			}
		}
	}

	allocation_size_byte = Utils::alignment(allocation_size_byte, sizeof(int));
	if(print_output)
		std::cout << "Number of Allocations: " << num_allocations << " | Allocation Size: " << allocation_size_byte << std::endl;

	int device{0};
	hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	
#ifdef TEST_CUDA
	std::cout << "--- CUDA ---\n";
	MemoryManagerCUDA memory_manager;
	std::string mem_name("CUDA");
#elif TEST_XMALLOC
	std::cout << "--- XMalloc ---\n";
	MemoryManagerXMalloc memory_manager;
	std::string mem_name("XMalloc");
#elif TEST_HALLOC
	std::cout << "--- Halloc ---\n";
	MemoryManagerHalloc memory_manager;
	std::string mem_name("Halloc");
#elif TEST_SCATTERALLOC
	std::cout << "--- ScatterAlloc ---\n";
	MemoryManagerScatterAlloc memory_manager;
	std::string mem_name("ScatterAlloc");
#elif TEST_OUROBOROS
	std::cout << "--- Ouroboros ---";
	#ifdef TEST_PAGES
	#ifdef TEST_VIRTUALIZED_ARRAY
	std::cout << " Page --- Virtualized Array ---\n";
	MemoryManagerOuroboros<OuroVAPQ> memory_manager;
	std::string mem_name("Ouroboros-P-VA");
	#elif TEST_VIRTUALIZED_LIST
	std::cout << " Page --- Virtualized List ---\n";
	MemoryManagerOuroboros<OuroVLPQ> memory_manager;
	std::string mem_name("Ouroboros-P-VL");
	#else
	std::cout << " Page --- Standard ---\n";
	MemoryManagerOuroboros<OuroPQ> memory_manager;
	std::string mem_name("Ouroboros-P-S");
	#endif
	#endif
	#ifdef TEST_CHUNKS
	#ifdef TEST_VIRTUALIZED_ARRAY
	std::cout << " Chunk --- Virtualized Array ---\n";
	MemoryManagerOuroboros<OuroVACQ> memory_manager;
	std::string mem_name("Ouroboros-C-VA");
	#elif TEST_VIRTUALIZED_LIST
	std::cout << " Chunk --- Virtualized List ---\n";
	MemoryManagerOuroboros<OuroVLCQ> memory_manager;
	std::string mem_name("Ouroboros-C-VL");
	#else
	std::cout << " Chunk --- Standard ---\n";
	MemoryManagerOuroboros<OuroCQ> memory_manager;
	std::string mem_name("Ouroboros-C-S");
	#endif
	#endif
#elif TEST_FDG
	std::cout << "--- FDGMalloc ---\n";
	MemoryManagerFDG memory_manager;
	std::string mem_name("FDGMalloc");
#elif TEST_REGEFF
	std::cout << "--- RegEff ---";
	#ifdef TEST_ATOMIC
	std::cout << " Atomic\n";
	MemoryManagerRegEff<RegEffVariants::AtomicMalloc> memory_manager;
	std::string mem_name("RegEff-A");
	#elif TEST_ATOMIC_WRAP
	std::cout << " Atomic Wrap\n";
	MemoryManagerRegEff<RegEffVariants::AWMalloc> memory_manager;
	std::string mem_name("RegEff-AW");
	#elif TEST_CIRCULAR
	std::cout << " Circular\n";
	MemoryManagerRegEff<RegEffVariants::CMalloc> memory_manager;
	std::string mem_name("RegEff-C");
	#elif TEST_CIRCULAR_FUSED
	std::cout << " Circular Fused\n";
	MemoryManagerRegEff<RegEffVariants::CFMalloc> memory_manager;
	std::string mem_name("RegEff-CF");
	#elif TEST_CIRCULAR_MULTI
	std::cout << " Circular Multi\n";
	MemoryManagerRegEff<RegEffVariants::CMMalloc> memory_manager;
	std::string mem_name("RegEff-CM");
	#elif TEST_CIRCULAR_FUSED_MULTI
	std::cout << " Circular Fused Multi\n";
	MemoryManagerRegEff<RegEffVariants::CFMMalloc> memory_manager;
	std::string mem_name("RegEff-CFM");
	#endif
#endif

	int** d_memory{nullptr};
	CHECK_ERROR(hipMalloc(&d_memory, sizeof(int*) * num_allocations));

	std::ofstream results_frag;
	results_frag.open((std::string("../results/frag_") + prop.name  + "_" + mem_name + "_" + std::to_string(num_allocations) + ".csv").c_str(), std::ios_base::app);
	results_frag << "\n" << allocation_size_byte << ",";

	int blockSize {256};
	int gridSize {Utils::divup<int>(num_allocations, blockSize)};

	for(auto i = 0; i < num_iterations; ++i)
	{
		if(warp_based)
			d_testAllocation <decltype(memory_manager), true> <<<gridSize * 32, blockSize>>>(memory_manager, d_memory, num_allocations, allocation_size_byte);
		else
			d_testAllocation <decltype(memory_manager), false> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations, allocation_size_byte);
		CHECK_ERROR(hipDeviceSynchronize());

		// Look at address range
		static int* static_min_ptr{reinterpret_cast<int*>(0xFFFFFFFFFFFFFFFFULL)};
		static int* static_max_ptr{nullptr};
		std::vector<int*> verification_pointers(num_allocations);
		CHECK_ERROR(hipMemcpy(verification_pointers.data(), d_memory, sizeof(int*) * verification_pointers.size(), hipMemcpyDeviceToHost));
		auto min_ptr = *min_element(verification_pointers.begin(), verification_pointers.end());
		auto max_ptr = *max_element(verification_pointers.begin(), verification_pointers.end());
		static_min_ptr = std::min(static_min_ptr, min_ptr);
		static_max_ptr = std::max(static_max_ptr, max_ptr);
		printf("%llu | %llu | %llu MB | %llu | %llu | %llu B\n", 
		reinterpret_cast<unsigned long long>(min_ptr), 
		reinterpret_cast<unsigned long long>(max_ptr), 
		(reinterpret_cast<unsigned long long>(max_ptr) - reinterpret_cast<unsigned long long>(min_ptr)) / (1024*1024),
		reinterpret_cast<unsigned long long>(static_min_ptr), 
		reinterpret_cast<unsigned long long>(static_max_ptr), 
		(reinterpret_cast<unsigned long long>(static_max_ptr) - reinterpret_cast<unsigned long long>(static_min_ptr)));
		results_frag << (reinterpret_cast<unsigned long long>(max_ptr) - reinterpret_cast<unsigned long long>(min_ptr)) 
			<< "," 
			<<(reinterpret_cast<unsigned long long>(static_max_ptr) - reinterpret_cast<unsigned long long>(static_min_ptr));
		if(num_iterations != 1)
			results_frag << ",";

		if(free_memory)
		{
			if(warp_based)
				d_testFree <decltype(memory_manager), true> <<<gridSize * 32, blockSize>>>(memory_manager, d_memory, num_allocations);
			else
				d_testFree <decltype(memory_manager), false> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations);
			CHECK_ERROR(hipDeviceSynchronize());
		}
	}
	
	return 0;
}