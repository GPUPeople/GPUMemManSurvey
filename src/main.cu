#include "hip/hip_runtime.h"
#include <iostream>

// ########################
#ifdef TEST_CUDA
#include "cuda/Instance.cuh"
#endif

__global__ void d_testFunctions(MemoryManagerCUDA memory_manager)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid > 0)
		return;

	int* test_array = reinterpret_cast<int*>(memory_manager.malloc(sizeof(int) * 16));

	for(int i = 0; i < 16; ++i)
	{
		test_array[i] = i;
	}

	memory_manager.free(test_array);

	printf("It worked!\n");

	return;
}

int main(int argc, char* argv[])
{
	std::cout << "Empty Testcase\n";

#ifdef TEST_CUDA
	MemoryManagerCUDA memory_manager;
#endif

	d_testFunctions <<<1,1>>>(memory_manager);

	hipDeviceSynchronize();

	printf("Testcase done!\n");

	return 0;
}