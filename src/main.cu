#include "hip/hip_runtime.h"
#include <iostream>

#include "UtilityFunctions.cuh"

// ########################
#ifdef TEST_CUDA
#include "cuda/Instance.cuh"
using MemoryManager = MemoryManagerCUDA;
const std::string mem_name("CUDA");
#elif TEST_HALLOC
#include "halloc/Instance.cuh"
using MemoryManager = MemoryManagerHalloc;
const std::string mem_name("HALLOC");
#elif TEST_XMALLOC
#include "xmalloc/Instance.cuh"
using MemoryManager = MemoryManagerXMalloc;
const std::string mem_name("XMALLOC");
#elif TEST_SCATTERALLOC
#include "scatteralloc/Instance.cuh"
using MemoryManager = MemoryManagerScatterAlloc;
const std::string mem_name("ScatterAlloc");
#elif TEST_FDG
#include "fdg/Instance.cuh"
using MemoryManager = MemoryManagerFDG;
const std::string mem_name("FDGMalloc");
#elif TEST_OUROBOROS
#include "ouroboros/Instance.cuh"
	#ifdef TEST_PAGES
	#ifdef TEST_VIRTUALIZED_ARRAY
	using MemoryManager = MemoryManagerOuroboros<OuroVAPQ>;
	const std::string mem_name("Ouroboros-P-VA");
	#elif TEST_VIRTUALIZED_LIST
	using MemoryManager = MemoryManagerOuroboros<OuroVLPQ>;
	const std::string mem_name("Ouroboros-P-VL");
	#else
	using MemoryManager = MemoryManagerOuroboros<OuroPQ>;
	const std::string mem_name("Ouroboros-P-S");
	#endif
	#endif
	#ifdef TEST_CHUNKS
	#ifdef TEST_VIRTUALIZED_ARRAY
	using MemoryManager = MemoryManagerOuroboros<OuroVACQ>;
	const std::string mem_name("Ouroboros-C-VA");
	#elif TEST_VIRTUALIZED_LIST
	using MemoryManager = MemoryManagerOuroboros<OuroVLCQ>;
	const std::string mem_name("Ouroboros-C-VL");
	#else
	using MemoryManager = MemoryManagerOuroboros<OuroCQ>;
	const std::string mem_name("Ouroboros-C-S");
	#endif
	#endif
#elif TEST_REGEFF
#include "regeff/Instance.cuh"
	#ifdef TEST_ATOMIC
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::AtomicMalloc>;
	const std::string mem_name("RegEff-A");
	#elif TEST_ATOMIC_WRAP
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::AWMalloc>;
	const std::string mem_name("RegEff-AW");
	#elif TEST_CIRCULAR
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CMalloc>;
	const std::string mem_name("RegEff-C");
	#elif TEST_CIRCULAR_FUSED
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CFMalloc>;
	const std::string mem_name("RegEff-CF");
	#elif TEST_CIRCULAR_MULTI
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CMMalloc>;
	const std::string mem_name("RegEff-CM");
	#elif TEST_CIRCULAR_FUSED_MULTI
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CFMMalloc>;
	const std::string mem_name("RegEff-CFM");
	#endif
#endif

template <typename MemoryManagerType>
__global__ void d_testAllocation(MemoryManagerType mm, int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;

	verification_ptr[tid] = reinterpret_cast<int*>(mm.malloc(allocation_size));
}

__global__ void d_testWriteToMemory(int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		ptr[i] = tid;
	}
}

__global__ void d_testReadFromMemory(int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		if(ptr[i] != tid)
		{
			printf("%d | We got a wrong value here! %d vs %d\n", tid, ptr[i], tid);
			__trap();
		}
	}
}

template <typename MemoryManagerType>
__global__ void d_testFree(MemoryManagerType mm, int** verification_ptr, int num_allocations)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;

	mm.free(verification_ptr[tid]);
}

int main(int argc, char* argv[])
{
	int device{0};
	hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << "\n";

	int num_allocations{10000};
	int allocation_size_byte{16};
	int num_iterations {10};
	if(argc >= 2)
	{
		num_allocations = atoi(argv[1]);
		if(argc >= 3)
		{
			allocation_size_byte = atoi(argv[2]);
			if(argc >= 4)
			{
				num_iterations = atoi(argv[3]);
			}
		}
	}
	allocation_size_byte = Utils::alignment(allocation_size_byte, sizeof(int));
	std::cout << "Number of Allocations: " << num_allocations << " | Allocation Size: " << allocation_size_byte << " | Iterations: " << num_iterations << std::endl;

	std::cout << "--- " << mem_name << "---\n";
	MemoryManager memory_manager(8192ULL * 1024ULL * 1024ULL);

	int** d_memory{nullptr};
	CHECK_ERROR(hipMalloc(&d_memory, sizeof(int*) * num_allocations));

	int blockSize {256};
	int gridSize {Utils::divup(num_allocations, blockSize)};
	float timing_allocation{0.0f};
	float timing_free{0.0f};
	hipEvent_t start, end;
	for(auto i = 0; i < num_iterations; ++i)
	{
		std::cout << "Iteration " << i + 1 << " / " << num_iterations << std::endl;
		Utils::start_clock(start, end);
		d_testAllocation <decltype(memory_manager)> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations, allocation_size_byte);
		timing_allocation += Utils::end_clock(start, end);

		CHECK_ERROR(hipDeviceSynchronize());

		d_testWriteToMemory<<<gridSize, blockSize>>>(d_memory, num_allocations, allocation_size_byte);

		CHECK_ERROR(hipDeviceSynchronize());

		d_testReadFromMemory<<<gridSize, blockSize>>>(d_memory, num_allocations, allocation_size_byte);

		CHECK_ERROR(hipDeviceSynchronize());

		Utils::start_clock(start, end);
		d_testFree <decltype(memory_manager)> <<<gridSize, blockSize>>>(memory_manager, d_memory, num_allocations);
		timing_free += Utils::end_clock(start, end);

		CHECK_ERROR(hipDeviceSynchronize());
	}
	timing_allocation /= num_iterations;
	timing_free /= num_iterations;

	std::cout << "Timing Allocation: " << timing_allocation << "ms" << std::endl;
	std::cout << "Timing       Free: " << timing_free << "ms" << std::endl;

	printf("Testcase done!\n");

	return 0;
}